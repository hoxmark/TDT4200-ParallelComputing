#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define BLOCKY 8
#define BLOCKX 8

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16
#define BLOCKDIM_X_SM20 32
#define BLOCKDIM_Y_SM20 32

#define PIXEL(i,j) ((i)*(j)*XSIZE)

/* Shorthand for less typing */
typedef unsigned char uchar;

/* Declarations of output functions */
void output(int* pixel);
void fancycolour(uchar *p, int iter);
void savebmp(char *name, uchar *buffer, int x, int y);

/* Struct for complex numbers */
typedef struct {
  float real, imag;
} complex_t;

/* Size of image, in pixels */
const int XSIZE = 2560;
const int YSIZE = 2048;

/* Max number of iterations */
const int MAXITER = 255;

/* Range in x direction */
const float xleft = -2.0;
const float xright = 1.0;
const float ycenter = 0.0;

/* Range in y direction, calculated in main
 * based on range in x direction and image size
 */
float yupper, ylower;

/* Distance between numbers */
float step;


/* Timing */
double walltime() {
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

/* Acutal GPU kenel which will be executed in parallel on the GPU */
__global__ void mandel_kernel( /* Add arguments here */ ){
      
}

__global__ void device_calculate(int *a, double xleft, double yupper, double step){
  int i = blockIdx.x * BLOCKX + threadIdx.x;
  int j = blockIdx.y * BLOCKY + threadIdx.y;

  complex_t c, z, temp;

  int iter = 0; 
  c.real = xleft + (step*i);
  c.imag = (yupper - step*j);

  z = c; 

  while (z.real*z.real + z.imag*z.imag<4.0){
    temp.real = z.real*z.real - z.imag*z.imag + c.real;
    temp.imag = 2.0 * z.real*z.imag + c.imag;
    z = temp;
    if (++iter == MAXITER) break;
  }

  a[PIXEL(i,j)]=iter;
  // a[PIXEL(i,j)]=1;
  

}


// cudaDeviceProp p;
//   cudaSetDevice(0);
//   cudaGetdeviceProperties(&p, 0);
//   printf("Device compute capanolity %d %d\n", p.major, p.minor)

/* Set up and call GPU kernel */
void calculate_cuda(int* pixel){
    // Allocate memory
    int *a; 
    hipMalloc((void**)&a, XSIZE*YSIZE* sizeof(int));
    // Compute thread-block size

    int gb = (XSIZE/BLOCKX)*(YSIZE/BLOCKY);
    int tb = BLOCKX*BLOCKY;
    // printf("%d\n", XSIZE/BLOCKX);
    // printf("%d\n", YSIZE/BLOCKY);
    // printf("%d\n", BLOCKX);
    // printf("%d\n", BLOCKY);
    // dim3 gridBlock = { XSIZE/BLOCKX, YSIZE/BLOCKY, 1};
    // dim3 gridBlock = { 1, 1, 1 };
    // dim3 threadBlock = {BLOCKX, BLOCKY, 1};
    // printf("%d", gridBlock);

    // Call kernel
    device_calculate<<<gb, tb>>>(a,xleft,yupper, step);

    printf("%s", hipGetErrorString(hipGetLastError()));

    // Transfer result from GPU to CPU
    hipMemcpy(pixel, a, XSIZE*YSIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(a);
}


// void host_calculate(){

// }
    


/* Calculate the number of iterations until divergence for each pixel.
 * If divergence never happens, return MAXITER
 */
void calculate(int* pixel) {
  for (int i = 0; i < XSIZE; i++) {
    for (int j = 0; j < YSIZE; j++) {
      complex_t c, z, temp;
      int iter = 0;
      c.real = (xleft + step * i);
      c.imag = (ylower + step * j);
      z = c;
      while (z.real * z.real + z.imag * z.imag < 4) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = 2 * z.real * z.imag + c.imag;
        z = temp;
        iter++;
        if(iter == MAXITER){
            break;
        }
      }
      pixel[j * XSIZE + i] = iter;
    }
  }
}


int main(int argc, char **argv) {
    
    /* Check input arguments */
  if (argc == 1) {
    puts("Usage: MANDEL n");
    puts("n decides whether image should be written to disk (1 = yes, 0 = no)");
    return 0;
  }
  
  /* Find number of CUDA devices (GPUs)
   * and print the name of the first one.
   */
  int n_devices;
  hipGetDeviceCount(&n_devices);
  printf("Number of CUDA devices: %d\n", n_devices);
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, 0);
  printf("CUDA device name: %s\n" , device_prop.name);
  
  /* Calculate the range in the y - axis such that we preserve the aspect ratio */
  step = (xright - xleft)/XSIZE;
  yupper = ycenter + (step * YSIZE)/2;
  ylower = ycenter - (step * YSIZE)/2;
  
  /* Global arrays for iteration counts/pixels
   * One array for the result of the CPU calculation,
   * one for the result of the GPU calculation.
   * (Both are in the host/CPU memory)
   */
  int* pixel_for_cpu = (int*) malloc(sizeof(int) * XSIZE * YSIZE);
  int* pixel_for_gpu = (int*) malloc(sizeof(int) * XSIZE * YSIZE);
  

  // /* Perform calculation on CPU */
  // double start_cpu = walltime();
  // calculate(pixel_for_cpu);
  // double end_cpu = walltime();
  
  /* Perform calculations on GPU */
  double start_gpu = walltime();
  calculate_cuda(pixel_for_gpu);

  double end_gpu = walltime();
  
  /* Compare execution times
   * The GPU time also includes the time for memory allocation and transfer
   */
  // printf("CPU time: %f s\n" , (end_cpu-start_cpu));
  printf("GPU time: %f s\n" , (end_gpu-start_gpu));
  

  /* Output */
  if (strtol(argv[1], NULL, 10) != 0) {
      output(pixel_for_gpu);
  }
  
  return 0;
}

/* Save 24 - bits bmp file, buffer must be in bmp format: upside - down */
void savebmp(char *name, uchar *buffer, int x, int y) {
  FILE *f = fopen(name, "wb");
  if (!f) {
    printf("Error writing image to disk.\n");
    return;
  }
  unsigned int size = x * y * 3 + 54;
  uchar header[54] = {'B', 'M',
                      size&255,
                      (size >> 8)&255,
                      (size >> 16)&255,
                      size >> 24,
                      0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, x&255, x >> 8, 0,
                      0, y&255, y >> 8, 0, 0, 1, 0, 24, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  fwrite(header, 1, 54, f);
  fwrite(buffer, 1, XSIZE * YSIZE * 3, f);
  fclose(f);
}

/* Given iteration number, set a colour */
void fancycolour(uchar *p, int iter) {
  if (iter == MAXITER);
  else if (iter < 8) { p[0] = 128 + iter * 16; p[1] = p[2] = 0; }
  else if (iter < 24) { p[0] = 255; p[1] = p[2] = (iter - 8) * 16; }
  else if (iter < 160) { p[0] = p[1] = 255 - (iter - 24) * 2; p[2] = 255; }
  else { p[0] = p[1] = (iter - 160) * 2; p[2] = 255 - (iter - 160) * 2; }
}

/* Create nice image from iteration counts. take care to create it upside down (bmp format) */
void output(int* pixel){
    unsigned char *buffer = (unsigned char*)calloc(XSIZE * YSIZE * 3, 1);
    for (int i = 0; i < XSIZE; i++) {
      for (int j = 0; j < YSIZE; j++) {
        int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
        fancycolour(buffer + p, pixel[(i + XSIZE * j)]);
      }
    }
    /* write image to disk */
    savebmp("mandel2.bmp", buffer, XSIZE, YSIZE);
    free(buffer);

}
